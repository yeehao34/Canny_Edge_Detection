#include "hip/hip_runtime.h"
﻿#include <iostream>
#include "opencv2/core/core.hpp"
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/imgproc/imgproc.hpp"
#include "cannyEdge.h"
#include <string>
#include <iomanip>
#include "hip/hip_runtime.h"
#include ""

int low_threshold = 20;
int high_threshold = 60;

const char *CW_IMG_ORIGINAL = "Original";
const char *CW_IMG_GRAY = "Grayscale";
const char *CW_IMG_EDGE = "Canny Edge Detection";

void doTransform(std::string, int);

__global__ void cuda_hello()
{
	printf("Hello World from GPU!\n");
}
int main(int argc, char **argv)
{
	cv::namedWindow(CW_IMG_ORIGINAL, cv::WINDOW_NORMAL);
	cv::namedWindow(CW_IMG_GRAY, cv::WINDOW_NORMAL);
	cv::namedWindow(CW_IMG_EDGE, cv::WINDOW_NORMAL);
	cv::resizeWindow(CW_IMG_ORIGINAL, 1280, 720);
	cv::resizeWindow(CW_IMG_GRAY, 1280, 720);
	cv::resizeWindow(CW_IMG_EDGE, 1280, 720);
	cv::moveWindow(CW_IMG_ORIGINAL, 10, 10);
	cv::moveWindow(CW_IMG_GRAY, 680, 10);
	cv::moveWindow(CW_IMG_EDGE, 1350, 10);

	int thd_per_blk;
	std::cout << "Please enter the number of threads per block :" << std::endl;
	std::cin >> thd_per_blk;

	// std::string img_path = "img/";
	int image_choice;
	std::cout << "       Image Dimensions      " << std::endl;
	std::cout << "#~~~~~~~~~~~~~~~~~~~~~~~~~~~~#" << std::endl;
	std::cout << "| " << std::left << std::setw(3) << "1." << std::setw(23) << "640x480.jpg"
			  << " |" << std::endl;
	std::cout << "| " << std::left << std::setw(3) << "2." << std::setw(23) << "1280x720.jpg"
			  << " |" << std::endl;
	std::cout << "| " << std::left << std::setw(3) << "3." << std::setw(23) << "1920x1080.jpg"
			  << " |" << std::endl;
	std::cout << "| " << std::left << std::setw(3) << "4." << std::setw(23) << "2560x1440.jpg"
			  << " |" << std::endl;
	std::cout << "| " << std::left << std::setw(3) << "5." << std::setw(23) << "3840x2160.jpg"
			  << " |" << std::endl;
	std::cout << "#~~~~~~~~~~~~~~~~~~~~~~~~~~~~#" << std::endl;
	std::cout << "Select image to do canny edge detection : " << std::endl;
	std::cin >> image_choice;
	std::string filepath = "";
	switch (image_choice)
	{
	case 1:
		filepath = "640x480";
		break;
	case 2:
		filepath = "1280x720";
		break;
	case 3:
		filepath = "1920x1080";
		break;
	case 4:
		filepath = "2560x1440";
		break;
	case 5:
		filepath = "3840x2160";
		break;
	default:
		break;
	}
	doTransform(filepath + ".jpg", thd_per_blk);

	cv::destroyAllWindows();
	return 0;
}

void doTransform(std::string file_path, int thd_per_blk)
{
	cv::Mat img_gray;
	std::string true_path = "../img/true/" + file_path;
	std::string save_path = "saved/" + file_path;

	cv::Mat img_ori = cv::imread("../img/" + file_path, 1);
	cv::cvtColor(img_ori, img_gray, cv::COLOR_BGR2GRAY);

	int w = img_gray.cols;
	int h = img_ori.rows;

	cv::Mat img_edge(h, w, CV_8UC1, cv::Scalar::all(0));
	apply_canny(img_edge.data, img_gray.data, low_threshold, high_threshold, w, h, thd_per_blk);

	cv::imwrite(save_path, img_edge);

	// Visualize all
	cv::imshow(CW_IMG_ORIGINAL, img_ori);
	cv::imshow(CW_IMG_GRAY, img_gray);
	cv::imshow(CW_IMG_EDGE, img_edge);

	char c = cv::waitKey(360000);
}
